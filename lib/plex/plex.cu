#include "hip/hip_runtime.h"
#include<stdio.h>
#include"plex.h"

__global__ void initTest()
{
    if(threadIdx.x == 0)
        printf("Hi There\n");
}

void plexTest()
{
    initTest<<<1,1>>>();
    hipDeviceSynchronize();
}